#include "hip/hip_runtime.h"
/**
 * Template for lab9, "Modern Computing Technologies"
 * 
 * Author: M.Grunwald

* When compiling on dwarf:
 *  use computing node: ssh61 - ssh67
 *  module load cuda/9.0 
 * 
 * source scl_source enable devtoolset-7 python27 
 * nvcc -arch sm_35 -O3 laplace2d.cu -o laplace2d -lcudart -lcufft -lm

 * */
#include <stdlib.h>
#include <stddef.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <thrust/complex.h>

#include "mct_utils.h"
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

// See for reference:
// https://en.cppreference.com/w/c/numeric/complex
#include <complex.h>



//=========================================================
// Functions
//=========================================================
/**
 * Test function
 * */
double function_xy(double x, double y)
{
#define A -0.03
#define B -0.01
#define C -0.005
    return exp(A * x * x + B * y * y + C * x * y);
}

/**
 * Analytical result
 * Use it for checking correctness!
 * */
double laplace_function_xy(double x, double y)
{
    double rdf2d_dx = function_xy(x, y) * (2. * A * x + C * y); // d/dx
    double rdf2d_dy = function_xy(x, y) * (2. * B * y + C * x); // d/dy

    double rlaplacef2d = rdf2d_dx * (2. * A * x + C * y) + function_xy(x, y) * (2. * A) + rdf2d_dy * (2. * B * y + C * x) + function_xy(x, y) * (2. * B); // laplace

    return rlaplacef2d;

#undef A
#undef B
#undef C
}

/**
 * You can use this function to check diff between two arrays
 * */
void test_array_diff(int N, double *a, double *b)
{
    int ixyz = 0;

    double d, d2;
    double maxd2 = 0.0;
    double sumd2 = 0.0;
    for (ixyz = 0; ixyz < N; ixyz++)
    {
        d = a[ixyz] - b[ixyz];

        d2 = d * d;
        sumd2 += d2;
        if (d2 > maxd2)
            maxd2 = d2;
    }

    printf("#    COMPARISON RESULTS:\n");
    printf("#           |max[a-b]| : %16.8g\n", sqrt(maxd2));
    printf("#         SUM[(a-b)^2] : %16.8g\n", sumd2);
    printf("# SQRT(SUM[(a-b)^2])/N : %16.8g\n", sqrt(sumd2) / N);
}

__global__ void calcFkxy(int nx, int ny, hipfftDoubleComplex* fk, hipfftDoubleComplex* fkxy)
{
    // Get our global thread ID
    size_t ixy = blockIdx.x*blockDim.x+threadIdx.x;
    hipfftDoubleComplex z;
    
    double Lx = 100.0; // width in x-direction
    double Ly = 100.0; // width in y-direction

    double dx = Lx/nx;
    double dy = Ly/ny;

    int ix, iy;
    double kx, ky, k2, factor;

    // Make sure we do not go out of bounds
    if(ixy < nx*ny){
        ix = ixy/ny;
        iy = ixy -ix*ny;

        if(ix<nx/2) kx=2.*M_PI/(dx*nx)*(ix);
        else kx=2.*M_PI/(dx*nx)*(ix-nx);
                
        if(iy<ny/2) ky=2.*M_PI/(dy*ny)*(iy);
        else ky=2.*M_PI/(dy*ny)*(iy-ny);

        k2 = -1.0*(kx*kx + ky*ky);
        factor = k2/(nx*2*(ny - 1)); //because here ny is "in reality" ny/2 + 1, so we have to turn it back into "real" ny, so 2*(ny-1) 

        z = fk[ixy];
        z.x*=factor;
        z.y*=factor;
        fkxy[ixy]=z;
    }

}

//=========================================================
// Main
//=========================================================

int main()
{

    //----------------------------------
    // Settings
    //----------------------------------
    bool debug = false;
    int nx = 4112; // number of points in x-direction
    int ny = 4112; // number of points in y-direction
    int blockSize=1024; // Number of threads in each thread block
    int gridSize; // Number of thread blocks in grid

    double Lx = 100.0; // width in x-direction
    double Ly = 100.0; // width in y-direction

    double x0 = -Lx / 2;
    double y0 = -Ly / 2;

    double dx = Lx/nx;
    double dy = Ly/ny;

    size_t sizeReal =  nx*ny*sizeof(double);
    size_t sizeComplex =  nx*(ny/2 +1)*sizeof(std::complex<double>);
 

    double *fxy_host; // function
    double *fxy_device;  
    std::complex<double> *fk_device; // transformed function
    std::complex<double> *fkxy_device;
    double *laplacefxy_host; // computed numerically
    double *laplacefxy_device;
    double *formula_laplacefxy; // computed according formula

    hipSetDevice(0); // to see devices use: nvidia-smi


    cppmallocl(fxy_host, nx*ny, double);
    hipMalloc(&fxy_device, sizeReal);
    hipMalloc(&fk_device, sizeComplex);
    hipMalloc(&fkxy_device, sizeComplex);
    cppmallocl(laplacefxy_host, nx*ny, double);
    hipMalloc(&laplacefxy_device, sizeReal);
    cppmallocl(formula_laplacefxy, nx*ny, double);

    //------------------------------------------------
    // Initialising function(s), copying to the device
    //------------------------------------------------

    int ix, iy, ixy=0;

    b_t(); //start timing (for initializing)

    for (ix = 0; ix < nx; ix++){
    	for (iy = 0; iy < ny; iy++){
	    // function
            fxy_host[ixy] = function_xy(x0 + dx * ix, y0 + dy * iy);

            // result for comparion
            formula_laplacefxy[ixy] = laplace_function_xy(x0 + dx * ix, y0 + dy * iy);
	    ixy++;
	}
    }
    double tInit = e_t(); // stop timing
    printf("# FUNCTIONS INITIALIZING TIME: %f sec\n", tInit);

    b_t(); //start timing (for sending)
    hipMemcpy(fxy_device, fxy_host, sizeReal, hipMemcpyHostToDevice);
    double tSend = e_t(); // stop timing
    printf("# COPYING (HOST->DEVICE) TIME: %f sec\n", tSend);

	//-------------------------------------------------------------------
	// preparing plans, transforming forward
	//--------------------------------------------------------------------
    hipfftHandle plan_f, plan_b;
    hipfftResult result;
    size_t required_size_f, required_size_b;

    b_t(); //start timing (for computing)
    result = hipfftPlan2d(&plan_f, nx, ny, HIPFFT_D2Z);
    if(debug)printf("Handle forward allocation - %d\n",(int) result);

    result = hipfftEstimate2d(nx, ny, HIPFFT_D2Z, &required_size_f);
    if(debug)printf("Worksize forward allocation - %d, size: %zd\n",(int) result, required_size_f);

    result = hipfftMakePlan2d(plan_f, nx, ny, HIPFFT_D2Z, &required_size_f);
    if(debug)printf("Plan forward making - %d\n",(int) result);

    result = hipfftExecD2Z(plan_f, (hipfftDoubleReal*)fxy_device, (hipfftDoubleComplex*)fk_device);
    if(debug)printf("Plan forward executing - %d\n",(int) result);

	//----------------------------------------------------------
	// creating transformed laplace
	//---------------------------------------------------------

    gridSize = (int)ceil((float)nx*(ny/2+1)/blockSize);
    calcFkxy<<<gridSize, blockSize>>>(nx, ny/2+1, (hipfftDoubleComplex*)fk_device, (hipfftDoubleComplex*)fkxy_device);

	//--------------------------------------------------
	// transforming backwards, coping to the host
	//--------------------------------------------------
    result = hipfftPlan2d(&plan_b, nx, ny, HIPFFT_Z2D);
    if(debug)printf("Handle backwards allocation - %d\n",(int) result);

    result = hipfftEstimate2d(nx, ny, HIPFFT_Z2D, &required_size_b);
    if(debug)printf("Worksize backwards allocation - %d, size: %zd\n",(int) result, required_size_b);

    result = hipfftMakePlan2d(plan_b, nx, ny, HIPFFT_Z2D, &required_size_b);
    if(debug)printf("Plan backwards making - %d\n",(int) result);

    result = hipfftExecZ2D(plan_b, (hipfftDoubleComplex*)fkxy_device, (hipfftDoubleReal*)laplacefxy_device);
    if(debug)printf("Plan backwards executing - %d\n",(int) result);

    double tCmpt = e_t(); // stop timing
    printf("# COMPUTING (FORWARD->KERNEL->BACKWARD) TIME: %f sec\n", tCmpt);

    b_t();//start timing (receive)
    hipMemcpy(laplacefxy_host, laplacefxy_device, sizeReal, hipMemcpyDeviceToHost);
    double tRecv = e_t(); // stop timing
    printf("# COPYING (DEVICE->HOST) TIME: %f sec\n", tRecv);
    printf("# TOTAL TIME: %f sec\n", tInit+tSend+tCmpt+tRecv);

	//--------------------------------------------------
	// checking results
	//--------------------------------------------------
    test_array_diff(nx * ny, laplacefxy_host, formula_laplacefxy);

    if(debug){
        ixy=0;
        for(ix=0; ix<nx; ix++){ 
            for(iy=0; iy<ny; iy++){    
                if(ix==nx/2) printf("%10.6g %10.6f %10.6f %10.6f\n", x0 + dx*ix, y0 + dy*iy, laplacefxy_host[ixy], formula_laplacefxy[ixy]);
                ixy++;
            }
        }        
    }


    hipFree(fxy_device);
    hipFree(fk_device);
    hipFree(fkxy_device);
    hipFree(laplacefxy_device);

    free(fxy_host);
    free(laplacefxy_host);
    free(formula_laplacefxy);

    return 1;
}
