#include "hip/hip_runtime.h"

/**
 * Task8, "Modern Computing Technologies"
 * 
 * Author: Jakub Zielinski
 * Date: 5.06.2022
 * 
 * nvcc -arch sm_35 -O3 laplace2d-cuda.cu -o laplace2d-cuda -lcudart -lcufft -lm
 * */

#include <stdlib.h>
#include <stddef.h>
#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../mct_utils.h"
#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#define BATCH 1

// See for reference:
// https://en.cppreference.com/w/c/numeric/complex
#include <complex.h>

// See for reference:
// https://docs.nvidia.com/cuda/cufft/index.html
#include <hipfft/hipfftw.h>

/**
 * Test function
 * */
double function_xy(double x, double y)
{
#define A -0.03
#define B -0.01
#define C -0.005
  return exp(A * x * x + B * y * y + C * x * y);
}

/**
 * Analytical result
 * Use it for checking correctness!
 * */
double laplace_function_xy(double x, double y)
{
  double rdf2d_dx = function_xy(x, y) * (2. * A * x + C * y); // d/dx
  double rdf2d_dy = function_xy(x, y) * (2. * B * y + C * x); // d/dy
  
  double rlaplacef2d = rdf2d_dx * (2. * A * x + C * y) + function_xy(x, y) * (2. * A) + rdf2d_dy * (2. * B * y + C * x) + function_xy(x, y) * (2. * B); // laplace
  
  return rlaplacef2d;

#undef A
#undef B
#undef C
}

/**
 * You can use this function to check diff between two arrays
 * */
void test_array_diff(int N, double *a, double *b)
{
  int ixyz = 0;
  
  double d, d2;
  double maxd2 = 0.0;
  double sumd2 = 0.0;
  for (ixyz = 0; ixyz < N; ixyz++){
    d = a[ixyz] - b[ixyz];
    
    d2 = d * d;
    sumd2 += d2;
    if (d2 > maxd2)
      maxd2 = d2;
  }

  printf("#    COMPARISON RESULTS:\n");
  printf("#           |max[a-b]| : %16.8g\n", sqrt(maxd2));
  printf("#         SUM[(a-b)^2] : %16.8g\n", sumd2);
  printf("# SQRT(SUM[(a-b)^2])/N : %16.8g\n", sqrt(sumd2) / N);
}


// CUDA kernels -------------------------->
__global__ void calculate_Fk(double dx, double dy, hipfftDoubleComplex* fK, int nx, int ny){
  hipfftDoubleComplex z;
  size_t ixy = blockIdx.x*blockDim.x+threadIdx.x;
  size_t ix = ixy/ny;
  size_t iy = ixy - ix*ny;
  double kx = 0;
  double ky = 0;
  if(ixy < nx && iy < ny/2+1){
    if(ix<nx/2) kx = 2.*M_PI/(dx*nx)*(ix   );
    else        kx = 2.*M_PI/(dx*nx)*(ix-nx);
      
    if(iy<ny/2) ky = 2.*M_PI/(dy*ny)*(iy   );
    else        ky = 2.*M_PI/(dy*ny)*(iy-ny);

    // recalculate fK real and imaginary part, as hipfftComplex  doesn't support some calculations 
    z=fK[ixy];
    z.x *= (-kx*kx - ky*ky) / (nx*ny);
    z.y *= (-kx*kx - ky*ky) / (nx*ny);
    fK[ixy] = z;
  }
}

//----------------------------------------<

int main()
{
  double GB = 1. /1024/1024/1024; // conversion to GB
  // plans for the FFTW
  hipfftHandle plan_f;
  hipfftHandle plan_b;
  
  // Settings
  int nx = 4112; // number of points in x-direction
  int ny = 4112; // number of points in y-direction	
  int n_r[2] = {nx,ny}; // dimensions for the fxy
  int n_c[2] = {nx,ny/2+1}; // dimensions for the fK
  
  double Lx = 100.0; // width in x-direction
  double Ly = 100.0; // width in y-direction
  
  double x0 = -Lx / 2;
  double y0 = -Ly / 2;
  
  double dx = Lx / nx;
  double dy = Ly / ny;
  
  double *h_fxy;
  double *laplacefxy;
  double *formula_laplacefxy;

  hipfftDoubleReal *fxy;                // function
  hipfftDoubleComplex *fK;              // array with f(kx, ky)

  double init_t, send_t, calc_t, recv_t; // for timing

  int ix, iy, ixy; // for iterating
  int blockSize, gridSize;

  hipSetDevice(0); // select GPU for CUDA
  
  //---- Allocate memory on the host ------>
  /* printf("Allocate memory\n"); */
  /* printf("\tAllocate memory for fxy at host\n"); */
  // hipHostMalloc((void**)&h_fxy, n_r[0] * n_r[1] * sizeof(double));
  cppmallocl(h_fxy, n_r[0] * n_r[1], double);
  /* printf("\tAllocate memory for laplacefxy at host\n"); */
  // hipHostMalloc((void**)&laplacefxy, n_r[0] * n_r[1] * sizeof(double));
  cppmallocl(laplacefxy, n_r[0] * n_r[1], double);
  /* printf("\tAllocate memory for fromula_laplacefxy at host\n"); */
  // hipHostMalloc((void**)&formula_laplacefxy, n_r[0] * n_r[1] * sizeof(double)); // this doesn't have to be coppied to CUDA
  cppmallocl(formula_laplacefxy, n_r[0] * n_r[1], double);
  //---------------------------------------<
  
  //---- Allocate memory on the device ---->
  /* printf("\tAllocate memory for fxy at device (cuda)\n"); */
  hipMalloc((void**)&fxy, n_r[0] * n_r[1] * sizeof(hipfftDoubleReal));
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return -1;
  }
  /* printf("\tAllocate memory for fK at device (cuda)\n"); */
  hipMalloc((void**)&fK, n_c[0] * n_c[1] * sizeof(hipfftDoubleComplex)); // this doesn't have to saved on the host
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return -1;
  }
  //---------------------------------------<
  
  //--- initialize vectors on the host ---->
  printf("Initializing the functions vectors\n");
  b_t();
  ixy = 0;
  for (ix = 0; ix < nx; ix++){
    for (iy = 0; iy < ny; iy++){
      // function
      h_fxy[ixy] = function_xy(x0 + dx * ix, y0 + dy * iy);
      
      // result for comparion
      formula_laplacefxy[ixy] = laplace_function_xy(x0 + dx * ix, y0 + dy * iy);
      ixy++;
    }
  }
  init_t = e_t();
  printf("Time for the initialization of the fxy: %lf s\n", init_t);
  //---------------------------------------<
  
  //---- Copy host vectors to device ------>
  b_t();
  hipMemcpy( fxy, h_fxy, sizeof(double)*n_r[0]*n_r[1], hipMemcpyHostToDevice);
  send_t = e_t();
  printf("Copy to device time: %f [sec]; bandwidth=%f [GB/sec]\n", send_t, sizeof(double)*n_r[0]*n_r[1]*GB/send_t);
  //---------------------------------------<

  /********* start the calculation ********/
  b_t();
  //------- create the forward plan ------->
  if(hipfftPlanMany(&plan_f, 2, n_r, NULL, 1, 0, NULL, 1, 0, HIPFFT_D2Z,BATCH) != HIPFFT_SUCCESS){
    fprintf(stderr,"CUFFT error: unable to create plan_f\n");
  }
  //--------------------------------------<
  
  //------- execute the forward plan ------>
  if(hipfftExecD2Z(plan_f, fxy, fK) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: unable to execute the forward transform\n");
  }//--------------------------------------<

  //------ calculate transform F(kx,ky) --->
  blockSize = 1024; // Number of threads in each thread block
  gridSize = (int)ceil((float)(n_c[0]*(n_c[1]/2+1))/blockSize); // Number of thread blocks in grid
  //---------------------------------------<
 
  // Execute the kernel
  calculate_Fk<<<gridSize, blockSize>>>(dx, dy, fK, nx, ny);

  //------ create the backward plan ------->
  if(hipfftPlanMany(&plan_b, 2, n_r, NULL, 1, 0, NULL, 1, 0, HIPFFT_Z2D,BATCH) != HIPFFT_SUCCESS){
    fprintf(stderr,"CUFFT error: unable to create backward plan\n");
  }
  //--------------------------------------<
  
  //----- execute the backward plan ------->
  if(hipfftExecZ2D(plan_b, fK, fxy) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: unable to execute the backward transform\n");
  }//--------------------------------------<
  /********* end of the calculation *******/
  calc_t = e_t();
  printf("Computation time: %lf s\n", calc_t);
    
  //------- Copy results to host ---------->
  b_t();
  hipMemcpy( laplacefxy, fxy, sizeof(double)*n_r[0]*n_r[1], hipMemcpyDeviceToHost);
  recv_t = e_t();
  printf("Copy to host time: %f [sec]; bandwidth=%f [GB/sec]\n", recv_t, sizeof(double)*nx*ny*GB/recv_t);
  //---------------------------------------<

  // Check correctness of computation
  test_array_diff(n_r[0] * n_r[1], laplacefxy, formula_laplacefxy);
  return 1;
}
